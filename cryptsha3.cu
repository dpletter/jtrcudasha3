#include "hip/hip_runtime.h"
/*
* This software is Copyright (c) 2013 Taylor Nelson/DJ Mitchell <eipeace2u at gmail dot com> <dj_trumpet at hotmail dot com> with inspiration from Lukas Odzioba's md5 implementaiton of the cuda format in john the ripper
* and it is hereby released to the general public under the following terms:
* Redistribution and use in source and binary forms, with or without modification, are permitted.
*/

//All the macros and constants
#include "../cuda_cryptsha3.h"

/**
 * cryptoState stores all of the input words, padded using the padding function
 */

__shared__ uint32_t cryptoState[NT][OW];

/*
 * Keccak state words.
 */
__shared__ uint64_t state[5*5*NT];

__global__ void testPadding (uint32_t *devInput, uint32_t *devOutput);
__device__ void padInputWord (uint32_t eval, uint32_t length);

// This is our padding function that pads with binary digits in the pattern 1(0)*1 until the input is 256 bits
// length is the number of characters in the input string
__device__ void padInputWord (uint32_t eval, uint32_t length)
{
	// Pointer to cryptoState word that we need to pad
 	// NOTE: this is a uint8_t NOT a uint32_t
	uint8_t *input = (uint8_t*) &cryptoState[eval][0];

	// Start at the end of this word and fill until we hit 32 characters
	uint32_t charIndex = length;

	input[charIndex++] = (1 << 7);

	// Go until index 30 and then fill it with zeroes
	while (charIndex < 31)
		input[charIndex++] = 0;
	
	// fill index 31 with 1
	input[charIndex] = 1;

}

__device__ void keccakBlockPermutation (uint32_t eval)
{
	uint32_t round, x, y;

	// Temporary storage.
	uint64_t C[5], D;

	// Linear feedback shift register for generating round constants.
	uint32_t LFSR = 1;

	// Get pointer to cryptoState for this evaluation.
	uint32_t *input = &cryptoState[eval][0];
	
	uint64_t tmp = 0;

	packAndReverseBytes(tmp, input[7], input[6]);
	state[index(0,0)] = tmp;
	packAndReverseBytes(tmp, input[5], input[4]);
	state[index(1,0)] = tmp;
	packAndReverseBytes(tmp, input[3], input[2]);
	state[index(2,0)] = tmp;
	packAndReverseBytes(tmp, input[1], input[0]);
	state[index(3,0)] = tmp;

	// Apply 24-round permutation.
	for (round = 0; round < 24; ++ round)
	{
		// Theta step.
		for (x = 0; x <= 4; ++ x)
		{
			C[x] = state[index(x,0)];
			for (y = 1; y <= 4; ++ y) 
				C[x] ^= state[index(x,y)];
		}
		for (x = 0; x <= 4; ++ x)
		{
			D = C[(x+4)%5] ^ ROT (C[(x+1)%5], 1);
			for (y = 0; y <= 4; ++ y)
			 	state[index(x,y)] ^= D;
		}

		// Rho step.
		// state[index(0,0)] = state[index(0,0)];
		state[index(1,0)] = ROT(state[index(1,0)], 1);
		state[index(0,2)] = ROT(state[index(0,2)], 3);
		state[index(2,1)] = ROT(state[index(2,1)], 6);
		state[index(1,2)] = ROT(state[index(1,2)], 10);
		state[index(2,3)] = ROT(state[index(2,3)], 15);
		state[index(3,3)] = ROT(state[index(3,3)], 21);
		state[index(3,0)] = ROT(state[index(3,0)], 28);
		state[index(0,1)] = ROT(state[index(0,1)], 36);
		state[index(1,3)] = ROT(state[index(1,3)], 45);
		state[index(3,1)] = ROT(state[index(3,1)], 55);
		state[index(1,4)] = ROT(state[index(1,4)], 2);
		state[index(4,4)] = ROT(state[index(4,4)], 14);
		state[index(4,0)] = ROT(state[index(4,0)], 27);
		state[index(0,3)] = ROT(state[index(0,3)], 41);
		state[index(3,4)] = ROT(state[index(3,4)], 56);
		state[index(4,3)] = ROT(state[index(4,3)], 8);
		state[index(3,2)] = ROT(state[index(3,2)], 25);
		state[index(2,2)] = ROT(state[index(2,2)], 43);
		state[index(2,0)] = ROT(state[index(2,0)], 62);
		state[index(0,4)] = ROT(state[index(0,4)], 18);
		state[index(4,2)] = ROT(state[index(4,2)], 39);
		state[index(2,4)] = ROT(state[index(2,4)], 61);
		state[index(4,1)] = ROT(state[index(4,1)], 20);
		state[index(1,1)] = ROT(state[index(1,1)], 44);

		// Pi step.
		// state[index(0,0)] = state[index(0,0)];
		D = state[index(1,3)];
		state[index(1,3)] = state[index(0,1)];
		state[index(0,1)] = state[index(3,0)];
		state[index(3,0)] = state[index(3,3)];
		state[index(3,3)] = state[index(2,3)];
		state[index(2,3)] = state[index(1,2)];
		state[index(1,2)] = state[index(2,1)];
		state[index(2,1)] = state[index(0,2)];
		state[index(0,2)] = state[index(1,0)];
		state[index(1,0)] = state[index(1,1)];
		state[index(1,1)] = state[index(4,1)];
		state[index(4,1)] = state[index(2,4)];
		state[index(2,4)] = state[index(4,2)];
		state[index(4,2)] = state[index(0,4)];
		state[index(0,4)] = state[index(2,0)];
		state[index(2,0)] = state[index(2,2)];
		state[index(2,2)] = state[index(3,2)];
		state[index(3,2)] = state[index(4,3)];
		state[index(4,3)] = state[index(3,4)];
		state[index(3,4)] = state[index(0,3)];
		state[index(0,3)] = state[index(4,0)];
		state[index(4,0)] = state[index(4,4)];
		state[index(4,4)] = state[index(1,4)];
		state[index(1,4)] = state[index(3,1)];
		state[index(3,1)] = D; // state[index(1,3)];

		// Chi step.
		for (y = 0; y <= 4; ++ y)
		{
			for (x = 0; x <= 4; ++ x)
				C[x] = state[index(x,y)] ^ ((~state[index((x+1)%5,y)]) &
					state[index((x+2)%5,y)]);
			for (x = 0; x <= 4; ++ x)
				state[index(x,y)] = C[x];
		}

		// Iota step.
		for (x = 0; x <= 6; ++ x)
		{
			state[index(0,0)] ^= (LFSR & 1ULL) << ((1 << x) - 1);
			LFSR = NEXT_STATE (LFSR);
		}
	}

	// Flip bytes back to Big-endian 32-bit words and put them into input
	tmp = state[index(0,0)];
	reverseBytesAndUnpack(tmp, input[7], input[6]);
	tmp = state[index(1,0)];
	reverseBytesAndUnpack(tmp, input[5], input[4]);
	tmp = state[index(2,0)];
	reverseBytesAndUnpack(tmp, input[3], input[2]);
	tmp = state[index(3,0)];
	reverseBytesAndUnpack(tmp, input[1], input[0]);

}

__global__ void keccakEntry (crypt_sha3_password *devInput, crypt_sha3_crack *devOutput, uint32_t L)
{
	uint32_t sample, eval;
	uint32_t maxIndex = 0;
	uint32_t temp = 0;

	// Sample number
	sample = blockIdx.y;
	sample *= gridDim.x;
	sample += blockIdx.x;
	sample *= blockDim.x;
	sample += threadIdx.x;

	// Proceed only if sample number is in bounds.
	// This is our boundary check
	if (sample < L)
	{
		// Evaluation number within block
		eval = sample % NT; 

		// Calculate number of 4byte words in this string
		maxIndex = (devInput[sample].length + 3) / 4;

		// Copy input word by word
		for (int i = 0; i < maxIndex; i++) {
			// Copy each byte of the word into a temp variable
			for (int j = 0; j < 4; j++) 
				temp |= ( devInput[sample].v[(i * 4) + j] >> (8 * j) ) ;
			
			cryptoState[eval][i] = temp;
			temp = 0;
		}

		// Use the padding function to pad the input to make it 256 bits
		padInputWord (eval, devInput[sample].length);

		for (int y = 0; y < 5; ++ y)
			for (int x = 0; x < 5; ++ x)
				state[index(x,y)] = 0;

		// Set Keccak state to 0 xor message block. Message block = input message
		// (32 bytes) plus padding of 10...01 (104 bytes), total = 136 bytes = 1088
		// bits. Little-endian byte orderin.
		keccakBlockPermutation (eval);

		// Store output.
		for (int i = 0; i < OW; ++ i) {
			devOutput[sample].hash[i * 4]     = 0xFF & (cryptoState[eval][i] >> 24);
			devOutput[sample].hash[i * 4 + 1] = 0xFF & (cryptoState[eval][i] >> 16);
			devOutput[sample].hash[i * 4 + 2] = 0xFF & (cryptoState[eval][i] >> 8);
			devOutput[sample].hash[i * 4 + 3] = 0xFF & cryptoState[eval][i];
		}
	}
}

/*
*	param inBuffer: input buffer of dictionary entries and their length in characters
*	param outBuffer: output buffer of hashes
*	param host_salt: This param is unused in this function and can probably be removed if salting is unnecessary
*	param L:  The number of entries in the dictionary passed into the function through inBuffer
*
*
*/

__host__ void sha3_crypt_gpu (crypt_sha3_password *inBuffer, crypt_sha3_crack *outBuffer, crypt_sha3_salt *host_salt, uint32_t L)
{
	//HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(cuda_salt), host_salt, sizeof(crypt_sha3_salt)));

	crypt_sha3_password *dev_inBuffer;
	crypt_sha3_crack *dev_outBuffer;

	size_t inSize = sizeof(crypt_sha3_password) * L;
	size_t outSize = sizeof(crypt_sha3_crack) * L;

	HANDLE_ERROR(hipMalloc((void**)&dev_inBuffer, inSize));
	HANDLE_ERROR(hipMalloc((void**)&dev_outBuffer, outSize));
	HANDLE_ERROR(hipMemcpy(dev_inBuffer, inBuffer, inSize, hipMemcpyHostToDevice));

	dim3 NB = dim3(X_BLOCKS, Y_BLOCKS);
	keccakEntry <<<NB, NT>>> (dev_inBuffer, dev_outBuffer, L);

	hipDeviceSynchronize();

	HANDLE_ERROR(hipMemcpy(outBuffer, dev_outBuffer, outSize, hipMemcpyDeviceToHost));

	HANDLE_ERROR(hipFree(dev_inBuffer));
	HANDLE_ERROR(hipFree(dev_outBuffer));
}
