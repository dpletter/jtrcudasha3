#include "hip/hip_runtime.h"
/*
* This software is Copyright (c) 2013 Taylor Nelson/DJ Mitchell <eipeace2u at gmail dot com> <dj_trumpet at hotmail dot com> with inspiration from Lukas Odzioba's md5 implementaiton of the cuda format in john the ripper
* and it is hereby released to the general public under the following terms:
* Redistribution and use in source and binary forms, with or without modification, are permitted.
*/

//All the macros and constants
#include "../cuda_cryptsha3.h"

/**
 * cryptoState stores all of the input words, padded using the padding function
 */

__shared__ uint32_t cryptoState[NT][OW];

/*
 * Keccak state words.
 */
__shared__ uint64_t state[5*5*NT];


// This is our padding function that pads with binary digits in the pattern 1(0)*1 until the input is 256 bits
__device__ void padInputWord (uint32_t eval, uint32_t length)
{
	// Pointer to cryptoState word that we need to pad
	uint8_t *input = &cryptoState[eval][0];

	// Start at the end of this word and fill until we hit 32 characters
	uint32_t charIndex = length;

	input[charIndex] = (1 << 7);

	// Go until index 30 and then fill it with zeroes
	while (charIndex < 31) 
		input[charIndex] = 0;
	
	// fill index 31 with 1
	input[charIndex] = 1;

}

__device__ void keccakBlockPermutation (uint32_t eval)
{
	uint32_t round, x, y;

	// Temporary storage.
	uint64_t C[5], D;

	// Linear feedback shift register for generating round constants.
	uint32_t LFSR = 1;

	// Get pointer to cryptoState for this evaluation.
	uint32_t *input = &cryptoState[eval][0];
	
	uint64_t tmp = 0;


	packAndReverseBytes (tmp, input[7], input[6]);
	state[index(0,0)] = tmp;
	packAndReverseBytes (tmp, input[5], input[4]);
	state[index(1,0)] = tmp;
	packAndReverseBytes (tmp, input[3], input[2]);
	state[index(2,0)] = tmp;
	packAndReverseBytes (tmp, input[1], input[0]);
	state[index(3,0)] = tmp;

	// Apply 24-round permutation.
	for (round = 0; round < 24; ++ round)
	{
		// Theta step.
		for (x = 0; x <= 4; ++ x)
		{
			C[x] = state[index(x,0)];
			for (y = 1; y <= 4; ++ y) 
				C[x] ^= state[index(x,y)];
		}
		for (x = 0; x <= 4; ++ x)
		{
			D = C[(x+4)%5] ^ ROT (C[(x+1)%5], 1);
			for (y = 0; y <= 4; ++ y)
			 	state[index(x,y)] ^= D;
		}

		// Rho step.
		// state[index(0,0)] = state[index(0,0)];
		state[index(1,0)] = ROT (state[index(1,0)],  1);
		state[index(0,2)] = ROT (state[index(0,2)],  3);
		state[index(2,1)] = ROT (state[index(2,1)],  6);
		state[index(1,2)] = ROT (state[index(1,2)], 10);
		state[index(2,3)] = ROT (state[index(2,3)], 15);
		state[index(3,3)] = ROT (state[index(3,3)], 21);
		state[index(3,0)] = ROT (state[index(3,0)], 28);
		state[index(0,1)] = ROT (state[index(0,1)], 36);
		state[index(1,3)] = ROT (state[index(1,3)], 45);
		state[index(3,1)] = ROT (state[index(3,1)], 55);
		state[index(1,4)] = ROT (state[index(1,4)],  2);
		state[index(4,4)] = ROT (state[index(4,4)], 14);
		state[index(4,0)] = ROT (state[index(4,0)], 27);
		state[index(0,3)] = ROT (state[index(0,3)], 41);
		state[index(3,4)] = ROT (state[index(3,4)], 56);
		state[index(4,3)] = ROT (state[index(4,3)],  8);
		state[index(3,2)] = ROT (state[index(3,2)], 25);
		state[index(2,2)] = ROT (state[index(2,2)], 43);
		state[index(2,0)] = ROT (state[index(2,0)], 62);
		state[index(0,4)] = ROT (state[index(0,4)], 18);
		state[index(4,2)] = ROT (state[index(4,2)], 39);
		state[index(2,4)] = ROT (state[index(2,4)], 61);
		state[index(4,1)] = ROT (state[index(4,1)], 20);
		state[index(1,1)] = ROT (state[index(1,1)], 44);

		// Pi step.
		// state[index(0,0)] = state[index(0,0)];
		D = state[index(1,3)];
		state[index(1,3)] = state[index(0,1)];
		state[index(0,1)] = state[index(3,0)];
		state[index(3,0)] = state[index(3,3)];
		state[index(3,3)] = state[index(2,3)];
		state[index(2,3)] = state[index(1,2)];
		state[index(1,2)] = state[index(2,1)];
		state[index(2,1)] = state[index(0,2)];
		state[index(0,2)] = state[index(1,0)];
		state[index(1,0)] = state[index(1,1)];
		state[index(1,1)] = state[index(4,1)];
		state[index(4,1)] = state[index(2,4)];
		state[index(2,4)] = state[index(4,2)];
		state[index(4,2)] = state[index(0,4)];
		state[index(0,4)] = state[index(2,0)];
		state[index(2,0)] = state[index(2,2)];
		state[index(2,2)] = state[index(3,2)];
		state[index(3,2)] = state[index(4,3)];
		state[index(4,3)] = state[index(3,4)];
		state[index(3,4)] = state[index(0,3)];
		state[index(0,3)] = state[index(4,0)];
		state[index(4,0)] = state[index(4,4)];
		state[index(4,4)] = state[index(1,4)];
		state[index(1,4)] = state[index(3,1)];
		state[index(3,1)] = D; // state[index(1,3)];

		// Chi step.
		for (y = 0; y <= 4; ++ y)
		{
			for (x = 0; x <= 4; ++ x)
				C[x] = state[index(x,y)] ^ ((~state[index((x+1)%5,y)]) &
					state[index((x+2)%5,y)]);
			for (x = 0; x <= 4; ++ x)
				state[index(x,y)] = C[x];
		}

		// Iota step.
		for (x = 0; x <= 6; ++ x)
		{
			state[index(0,0)] ^= (LFSR & 1ULL) << ((1 << x) - 1);
			LFSR = NEXT_STATE (LFSR);
		}
	}

	// Flip bytes back to Big-endian 32-bit words and put them into input
	tmp = state[index(0,0)];
	reverseBytesAndUnpack (tmp, input[7], input[6]);
	tmp = state[index(1,0)];
	reverseBytesAndUnpack (tmp, input[5], input[4]);
	tmp = state[index(2,0)];
	reverseBytesAndUnpack (tmp, input[3], input[2]);
	tmp = state[index(3,0)];
	reverseBytesAndUnpack (tmp, input[1], input[0]);

}

__global__ void keccakEntry (crypt_sha3_password *devInput, crypt_sha3_crack *devOutput, uint32_t trial, uint32_t L)
{
	uint32_t sample, eval;

	// Sample number
	sample = blockIdx.y;  
	sample *= gridDim.x;
	sample += blockIdx.x;
	sample *= blockDim.x;
	sample += threadIdx.x;

	// Proceed only if sample number is in bounds.
	// This is our boundary check
	if (sample < L)
	{
		// Evaluation number within block
		eval = sample % NT; 

		// Read input from devInput
		cryptoState[eval] = &devInput[sample].v;

		// Use the padding function to pad the input to make it 256 bits
		padInputWord (eval, devInput[sample].length);

		// Set Keccak state to 0 xor message block. Message block = input message
		// (32 bytes) plus padding of 10...01 (104 bytes), total = 136 bytes = 1088
		// bits. Little-endian byte orderin.

		// memset is way cleaner??
		for (y = 0; y < 5; ++ y)
			for (x = 0; x < 5; ++ x)
				state[index(x,y)] = 0;

		// Compute crypto function.
		// Do we even need to pass in the word here? 

		// xor in new state with first word in wordlen
		// for (int ctr = 0; ctr < wordlen; ctr++) 
		// is this index calculation correct?
		keccakBlockPermutation (eval);

			// Store output.
		for (int i = 0; i < OW; ++ i)
			devOutput[sample*OW + i] = cryptoState[eval][i];


	}
}

__host__ void sha3_crypt_gpu (crypt_sha3_password *inBuffer, crypt_sha3_crack *outBuffer, crypt_sha3_salt *host_salt, uint32_t L)
{
	HANDLE_ERROR(hipMemcpyToSymbol(HIP_SYMBOL(cuda_salt), host_salt, sizeof(crypt_sha3_salt)));

	crypt_sha3_password *dev_inBuffer;
	crypt_sha3_crack *dev_outBuffer;

	size_t inSize = sizeof(crypt_sha3_password) * KEYS_PER_CRYPT;
	size_t outSize = sizeof(crypt_sha3_crack) * KEYS_PER_CRYPT;

	HANDLE_ERROR(hipMalloc(&dev_inBuffer, inSize));
	HANDLE_ERROR(hipMalloc(*dev_outBuffer, inSize));
	HANDLE_ERROR(hipMemcpy(dev_inBuffer, inBuffer, inSize, hipMemcpyHostToDevice));

	// Double check my math on this calculation of number of blocks
	keccakEntry <<<((L + NT - 1) / NT) , NT>>> (dev_inBuffer, dev_outBuffer, 0, L);

	HANDLE_ERROR(hipMemcpy(outBuffer, dev_inBuffer, outSize, hipMemcpyDeviceToHost));

	HANDLE_ERROR(hipFree(dev_inBuffer));
	HANDLE_ERROR(hipFree(dev_outBuffer));
}


